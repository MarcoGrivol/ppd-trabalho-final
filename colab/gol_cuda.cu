#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <sys/time.h>

__global__ void updateState(bool *d_grid, bool *d_next_grid, int N) {
    // gets the row and col from thread and block index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        int sum =  
            d_grid[(i - 1) * N + j - 1] + // [-1, -1]
            d_grid[(i - 1) * N + j    ] + // [-1,  0]
            d_grid[(i - 1) * N + j + 1] + // [-1,  1]
            d_grid[ i      * N + j - 1] + // [ 0, -1]
            d_grid[ i      * N + j + 1] + // [ 0,  1]
            d_grid[(i + 1) * N + j - 1] + // [ 1, -1]
            d_grid[(i + 1) * N + j    ] + // [ 1,  0]
            d_grid[(i + 1) * N + j + 1];  // [ 1,  1]
        if (d_grid[i * N + j]) {
            // is alive
            if (sum == 2 || sum == 3) d_next_grid[i * N + j] = true;
            else d_next_grid[i * N + j] = false;
        } 
        else {
            // is dead
            if (sum == 3) d_next_grid[i * N + j] = true;
            else d_next_grid[i * N + j] = false;
        }
    }
    return;
}

__global__ void copyBordersUpDown(bool *d_grid, bool *d_next_grid, int N) {
    // wrap-around padding
    // first and last row 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col > 0 && col < N - 1) {
        d_next_grid[col] = d_next_grid[(N - 2) * N + col];
        d_next_grid[(N - 1) * N + col] = d_next_grid[N + col];
    }
    return;
}

__global__ void copyBordersLeftRight(bool *d_grid, bool *d_next_grid, int N) {
    // wrap-around padding
    // left and right columns
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (row >= 0 && row < N) {
        d_next_grid[row * N] = d_next_grid[row * N + N - 2];
        d_next_grid[row * N + N - 1] = d_next_grid[row * N + 1];
    }
    return;
}

void swap(bool **d_grid, bool **d_next_grid) {
    // Update grid to receive last iteration
    bool *aux = *d_grid;
    *d_grid = *d_next_grid;
    *d_next_grid = aux;
}

void runAllGenerations(int N, int G, bool *d_grid, bool *d_next_grid) {

    // defines the # of threads required
    int threads = 32; // 32 * 32 = 1024 -> max # of threads per block
    dim3 blocks (ceil((float) N / threads), ceil((float) N / threads)); // blocks in grid
    dim3 threads_per_block (threads, threads);
    printf("Threads: %d\n", 
        threads_per_block.x * threads_per_block.y * blocks.x * blocks.y
    );
    printf("blocks (%d, %d)\nthreads_per_block (%d, %d)\n", 
        blocks.x, blocks.y,
        threads_per_block.x, threads_per_block.y
    );

    // Updates the state of the grid G times
    for (int i = 0; i < G; i++) {
        updateState<<<blocks, threads_per_block>>>(d_grid, d_next_grid, N);
        copyBordersUpDown<<<blocks, threads_per_block>>>(d_grid, d_next_grid, N);
        copyBordersLeftRight<<<blocks, threads_per_block>>>(d_grid, d_next_grid, N);
        swap(&d_grid, &d_next_grid);
    }
}

int main(int argc, char *argv[]) {
    srand(42);
    
    if (argc != 3) {
        printf("gol_sequential.c requires N and G integers.\n");
        return -1;
    }
    int N = atoi(argv[1]) + 2; // size of grid (N x N)
    int G = atoi(argv[2]);     // # of generations

    bool *grid = (bool *) malloc(N * N * sizeof(bool));
    bool *next_grid = (bool *) malloc(N * N * sizeof(bool));
    // randomly initialize grid
    for (int i = 1; i < N - 1; i++) {
        for (int j = 1; j < N - 1; j++) {
            grid[i * N + j] = (int) rand() % 2;
        }
    }
    // wrap-around padding
    // copy first and last row borders
    for (int col = 1; col < N - 1; col++) {
        grid[col] = grid[(N - 2) * N + col];
        grid[(N - 1) * N + col] = grid[N + col];
    }
    // copy left and right borders
    for (int row = 0; row < N; row++) {
        grid[row * N] = grid[row * N + N - 2];
        grid[row * N + N - 1] = grid[row * N + 1];
    }

    hipEvent_t t0, t1;
    hipEventCreate(&t0);
    hipEventCreate(&t1);
    // start timer
    hipEventRecord(t0);

    // device memory allocation
    bool *d_grid, *d_next_grid;
    hipMalloc(&d_grid, N * N * sizeof(bool));
    hipMalloc(&d_next_grid, N * N * sizeof(bool));
    // copy grid to device
    hipMemcpy(d_grid, grid, N * N * sizeof(bool), hipMemcpyHostToDevice);

    runAllGenerations(N, G, d_grid, d_next_grid);

    // copy grid from device to host
    hipMemcpy(grid, d_next_grid, N * N * sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_grid);
    hipFree(d_next_grid);

    hipEventRecord(t1);
    // end timer

    hipEventSynchronize(t1);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, t0, t1);
    elapsed_time /= 1000.;
    printf("Time for grid %d x %d and %d generations: %.3f s\n", N-2, N-2, G, elapsed_time);
    printf("Generations per second: %.1f g/s\n", (float) G / elapsed_time);
    //verify integrity
    bool *correct_grid = (bool *) malloc((N - 2) * (N - 2) * sizeof(bool));
    FILE *fp;
    fp = fopen("correct_grid.bin", "rb");
    int rc = fread(correct_grid, sizeof(*correct_grid), (N - 2) * (N - 2), fp);
    fclose(fp);
    if (rc == (N - 2) * (N - 2)) {
        int errors = 0;
        for (int i = 1; i < N - 1; i++) {
            for (int j = 1; j < N - 1; j++) {
                if (correct_grid[(i - 1) * (N - 2) + j - 1] != grid[i * N + j]) {
                    errors++;
                }
            }
        }
        printf("%d errors\n", errors);
    }
    else {
        printf("ERROR in reading file\n");
        printf("Successfully read %d elements\n", rc);
    }
    
    free(grid);
    free(next_grid);
    return 0;
}